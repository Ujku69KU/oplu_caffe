#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/oplu_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void OPLUForward(const int n, const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP2(index, n) {
    if (in[index] > in[index+1]) {
       out[index] = in[index];
       out[index+1] = in[index+1];
     }
    else {
       out[index+1] = in[index];
       out[index] = in[index+1];  
    }
  }
}


template <typename Dtype>
void OPLULayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();

  // NOLINT_NEXT_LINE(whitespace/operators)
  OPLUForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data);
  CUDA_POST_KERNEL_CHECK;
}


template <typename Dtype>
__global__ void OPLUBackward(const int n, const Dtype* in_diff,
    const Dtype* out_data, const Dtype* in_data,
    Dtype* out_diff) {

    CUDA_KERNEL_LOOP2(index, n) {
    if (in_data[index] > in_data[index+1]) {
       out_diff[index] = in_diff[index];
       out_diff[index+1] = in_diff[index+1];
     }
    else {
       out_diff[index+1] = in_diff[index];
       out_diff[index] = in_diff[index+1];  
    }
 }
}


template <typename Dtype>
void OPLULayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    const Dtype* top_data = top[0]->gpu_data();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();

    // NOLINT_NEXT_LINE(whitespace/operators)
    OPLUBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, top_data, bottom_data, bottom_diff);
    CUDA_POST_KERNEL_CHECK;
  }
}


INSTANTIATE_LAYER_GPU_FUNCS(OPLULayer);


}  // namespace caffe
